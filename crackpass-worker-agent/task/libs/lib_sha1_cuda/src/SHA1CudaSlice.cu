#include "hip/hip_runtime.h"
#include <iostream>

#include <cstdio>
#include <cstdlib>

#include "SHA1Cuda.hpp"
#include "SHA1CudaSlice.hpp"
#include "kernel/SHA1CudaKernel.hpp"


//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
// => For debugging purposes
//

//@huypn
/*
   Catching Cuda Errors
 */
static void HandleError(
        hipError_t err,
        const char *file,
        int line
        )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/**
  printing base
  */
#ifdef _DEBUG
void printBase(
        std::vector<unsigned char> base,
        std::string charset,
        size_t baseLen
        )
{
    std::cout << "szBase = ";
    for( int i = 0; i < baseLen; i++ )
    {
        std::cout << charset[base[i]];
    }
    std::cout << std::endl;
}
#endif


//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

SHA1CudaSlice::SHA1CudaSlice(
        SHA1Cuda *controller
        )
{
    fController = controller;
    fSliceState = SLICE_STATE_READY;

    // init cuda params
    fDeviceId = -1;
    fBlockSize = 0;
    fGridSize = 0;
    fStreamCount = 0;
    fOffset = 0;

    //@huypn: copying running parameters
    fController->CopyBasicParams( fHashDigest, fCharsetStr );
#ifdef _DEBUG
    // DEBUG: print out digest
#endif

    // Taking care of charset
    fCharsetLen = fCharsetStr.size();
    for( int i = 0; i < fCharsetLen; i++ ) {
        fCharsetUChar.push_back( (unsigned char) fCharsetStr[i] );
    }
#ifdef _DEBUG
    for( int i = 0; i < fCharsetLen; i++ ) {
        std::cout << fCharsetUChar[i];
    }
    std::cout << std::endl;
#endif

    fInputBase.resize(16);
    std::fill(fInputBase.begin(), fInputBase.end(), 0);

}


SHA1CudaSlice::~SHA1CudaSlice()
{
    // Finalize work
    StopSlice();
    delete fWorkerThread;
}


void SHA1CudaSlice::FreeCuda() {
    // Sync device
    hipDeviceSynchronize();
    // Destroy texture object
    hipDestroyTextureObject(texBase);
    hipDestroyTextureObject(texCharset);
    // Free CUDA malloc 
    hipFree(d_kernelRes);
    hipFree(d_charset);
    hipFree(d_base);
    // Reset Cuda Device
    hipDeviceReset();
}

void SHA1CudaSlice::StartSlice( int deviceId )
{
    fSliceState = SLICE_STATE_RUNNING;
    fDeviceId = deviceId;
    fWorkerThread = new boost::thread( boost::bind(&SHA1CudaSlice::WorkingLoop, this) );
}

void SHA1CudaSlice::StopSlice()
{
    fSliceState = SLICE_STATE_STOPPED;
    hipDeviceSynchronize();
    fWorkerThread->join();
}


void SHA1CudaSlice::Init()
{
    HANDLE_ERROR( hipSetDevice( fDeviceId ) );
    HANDLE_ERROR( hipGetDeviceProperties( &fDeviceProps, fDeviceId ));

    //#CUDA_OCCUPANCY_API: Get blocksize and gridsize
    int blockSize = 0;
    int minGridSize = 0;
    HANDLE_ERROR( hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)SHA1CudaKernel, 0, 0) );
    HANDLE_ERROR( hipDeviceSynchronize() );
    std::cout << "OCCUPANCY APIs: blocksize=" << blockSize << ";gridSize=" << minGridSize << std::endl;
    fBlockSize = blockSize;
    fGridSize = minGridSize;
    fStreamCount = 1;
    fOffset = fBlockSize * fGridSize * fStreamCount;

    // emergency flag: set when result found;
    h_kernelRes = (uint4 *) malloc(sizeof(uint4));
    *h_kernelRes = make_uint4(0,0,0,0);
    HANDLE_ERROR( hipMalloc((void**)&d_kernelRes, sizeof(uint4)) );
    HANDLE_ERROR( hipMemcpy(d_kernelRes, h_kernelRes, sizeof(uint4),\
                hipMemcpyHostToDevice) );

    // #CUDA_BINDLESS_TEXTURE
    // SHA1, digest does not fit any vector types -> load in to texture
    HANDLE_ERROR( hipMalloc((void**) &d_digest, 20 * sizeof(unsigned char)) );
    HANDLE_ERROR( hipMemcpy(d_digest, &fHashDigest[0], 20 * sizeof(unsigned char),\
                hipMemcpyHostToDevice) );
    hipResourceDesc resDescDigest;
    memset( &resDescDigest, 0, sizeof(resDescDigest) );
    resDescDigest.resType = hipResourceTypeLinear;
    resDescDigest.res.linear.devPtr = d_digest;
    resDescDigest.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDescDigest.res.linear.desc.x = 8;
    resDescDigest.res.linear.sizeInBytes = 20 * sizeof(unsigned char);
    hipTextureDesc texDescDigest;
    memset(&texDescDigest, 0, sizeof(texDescDigest));
    texDescDigest.readMode = hipReadModeElementType;
    texDigest = 0;
    HANDLE_ERROR( hipCreateTextureObject(&texDigest, &resDescDigest, &texDescDigest, NULL) );

    // Allocate input base, load into mapped memory
    HANDLE_ERROR( hipMalloc( (void**)&d_base, 16*sizeof(unsigned char)) );
    HANDLE_ERROR( hipMemcpy( d_base, &fInputBase[0], 16*sizeof(unsigned char),\
                hipMemcpyHostToDevice) );
    hipResourceDesc resDescBase;
    memset( &resDescBase, 0, sizeof(resDescBase) );
    resDescBase.resType = hipResourceTypeLinear;
    resDescBase.res.linear.devPtr = d_base;
    resDescBase.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDescBase.res.linear.desc.x = 8;
    resDescBase.res.linear.sizeInBytes = 16 * sizeof( unsigned char );
    hipTextureDesc texDescBase;
    memset(&texDescBase, 0, sizeof(texDescBase));
    texDescBase.readMode = hipReadModeElementType;
    texBase = 0;
    HANDLE_ERROR( hipCreateTextureObject( &texBase, &resDescBase, &texDescBase, NULL ) );

    // Allocate charset; load into bindless texture 
    HANDLE_ERROR( hipMalloc((void**)&d_charset, sizeof(unsigned char)*fCharsetLen) );
    HANDLE_ERROR( hipMemcpy(d_charset, &fCharsetUChar[0], sizeof(unsigned char)*fCharsetLen, hipMemcpyHostToDevice) );
    hipResourceDesc resDescCharset;
    memset( &resDescCharset, 0, sizeof(resDescCharset) );
    resDescCharset.resType = hipResourceTypeLinear;
    resDescCharset.res.linear.devPtr = d_charset;
    resDescCharset.res.linear.desc.f = hipChannelFormatKindUnsigned;
    resDescCharset.res.linear.desc.x = 8;
    resDescCharset.res.linear.sizeInBytes = fCharsetLen*sizeof( unsigned char );
    hipTextureDesc texDescCharset;
    memset(&texDescCharset, 0, sizeof(texDescCharset));
    texDescCharset.readMode = hipReadModeElementType;
    texCharset=0;
    HANDLE_ERROR( hipCreateTextureObject( &texCharset, &resDescCharset, &texDescCharset, NULL ) );
}


// Kernel wrapper
int SHA1CudaSlice::CallKernel( int worksize )
{
    // Setting up launching parameter
    int blockSize = fBlockSize;
    int gridSize = (worksize + blockSize - 1) / blockSize;
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(gridSize, 1, 1);
    SHA1CudaKernel<<<dimGrid, dimBlock>>>(
            texDigest, d_kernelRes,
            texBase, fInputLen,
            texCharset , fCharsetLen,
            worksize
            );
    HANDLE_ERROR( hipDeviceSynchronize() );
    int ret = -1;
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        ret = -2;
    } else {
        ret = -1;
        hipMemcpy(h_kernelRes, d_kernelRes, sizeof(uint4), hipMemcpyDeviceToHost);
        if( h_kernelRes->y == 1 ) {
             ret = 0;
        } 
    }
    return ret;
}


void SHA1CudaSlice::WorkingLoop()
{
    Init();
    while( fSliceState != SLICE_STATE_STOPPED &&
            fSliceState != SLICE_STATE_FINISHED ) {
        // Prepare data.
        int requestOffset = fOffset;
        bool hasData = fController->PopRequest(
                fInputBase, fInputLen, requestOffset );
        if (!hasData) {
            // Avoid CPU consuming when idle;
            boost::this_thread::sleep(
                    boost::posix_time::milliseconds(50));
            continue;
        }
        // Update base for new kernel call
        HANDLE_ERROR( hipMemcpy( d_base, &fInputBase[0], 16*sizeof(unsigned char), hipMemcpyHostToDevice) );
        // Call kernel
        int ret = CallKernel(requestOffset);
        // CUDA error -> dung pha ma
        if( ret == -2 ) {
            FreeCuda();
            return;
        } else if( ret != -1 ) {
            uint32_t counter = h_kernelRes->x;
            uint32_t i = fInputLen;
            for (uint32_t j = 0, a = 0, carry = 0;
                    j < i;
                    ++j, counter /= fCharsetLen)
            {
                a = fInputBase[j] + carry + counter % fCharsetLen;
                if (a >= fCharsetLen) {
                    carry = 1;
                    a -= fCharsetLen;
                }
                else carry = 0;
                fResultStr.push_back( fCharsetStr[a] );
            }
            fController->PushResult( fResultStr );
            std::cout << "Found result: " << fResultStr <<std::endl;
            return;
        }
    }
}



